/*
* IT 388
* Omar Baloch
* Homework 5
* Matrix addition
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>

/* Cuda function that adds two Matrices */ 
__global__ void addMatrices(double *x, double *y, double *z, int n, int m){
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int column = blockIdx.y * blockDim.y + threadIdx.y;
  int index = row*m +column;
  if(row < n && column < m){
    z[index] = x[index] + y[index];
  }
}

/*--------------------------------
 Function to Print Matrices
 --------------------------------*/
void printMatrix(double *x,double *y,double *c, int n, int m) {
    int i,j;
    for(i=0;i<n;i++){
        for(j=0;j<m;j++){
            printf("%.1f ", x[i*n+j]);
        }
        printf("\n");
    }
    printf("+\n");
    for(i=0;i<n;i++){
        for(j=0;j<m;j++){
            printf("%.1f ", y[i*n+j]);
        }
        printf("\n");
    }
    printf("=\n");
    for(i=0;i<n;i++){
        for(j=0;j<m;j++){
            printf("%.1f ", c[i*n+j]);
        }
        printf("\n");
    }
}

/*--------------------------------
 Function to measure run time
 --------------------------------*/
double getTime(){
    struct timeval t;
    gettimeofday(&t, NULL);
    return t.tv_sec + t.tv_usec/1000000.0;

}

/*------------------------------------------------------------------*/
int main(int argc, char* argv[]) {

    double start, elapsed;
    double* z; double* x; double* y;
    int n,m;
    //int blockSize, numBlocks;
    if(argv[1] == 0 || argv[2] == 0 || argv[3] == 0 || argv[4] == 0){
        fprintf(stderr,"\n \t USAGE: <N> <M> <Block Size (x)> <Block Size (y)>\n\n");
        exit(1);
    }
    n=strtol(argv[1], NULL, 10); 
    m=strtol(argv[2], NULL, 10);
    int size = n * m;
    int blockSizeX = strtol(argv[3], NULL, 10);
    int blockSizeY = strtol(argv[4], NULL, 10);
    int numBlocksX = ceil(double(n) / blockSizeX);
    dim3 blockSize=(blockSizeX,blockSizeY);
    dim3 numBlocks = (numBlocksX,1);
    int i,j;
    hipMallocManaged(&y, size*sizeof(double));
    hipMallocManaged(&x, size*sizeof(double));
    hipMallocManaged(&z, size*sizeof(double));

    // generate matrixes
    for (i=0;i<n;++i){
        for(j=0;j<m;++j){
            x[i*m+j]=sin(i*m+j)*sin(i*m+j);
            y[i*m+j]=cos(i*m+j)*cos(i*m+j);

        }
    }
    start = getTime();
    addMatrices<<<numBlocks,blockSize>>>(x, y, z, n, m);
    hipDeviceSynchronize();
    elapsed = getTime() - start;
    // Print matrices
    if (n<20 && m<20){
        printMatrix(x,y,z,n, m);
    }
    
    printf("Number of blocks: %d, Block size x-direction: %d, Block size y-direction: %d, Matrix size(NxM): %dx%d, Elapsed time:%f\n",numBlocksX,blockSizeX,blockSizeY,n,m,elapsed);
    hipFree(x); hipFree(y); hipFree(z);
    return 0;
}  /* main */